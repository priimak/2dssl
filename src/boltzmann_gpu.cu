#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <gsl/gsl_specfunc.h>
#include "boltzmann.h"

#define PPP 64

extern "C"
void HandleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString( err ),
           file, line );
    exit( EXIT_FAILURE );
  }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

extern ffloat host_E_dc, host_E_omega, host_omega, host_mu, host_alpha,
  PhiYmin, PhiYmax, host_B, t_start, host_dPhi, host_dt,
  host_bdt, host_nu_tilde, host_nu2, host_nu;

extern int host_M, host_N, MSIZE, MP1, NSIZE, host_TMSIZE;

__constant__ ffloat E_dc, E_omega, omega, B, dt, dPhi, nu, nu2, nu_tilde, bdt, mu, alpha, dev_PhiYmin;
__constant__ int M, N, dev_MSIZE, TMSIZE, dev_NSIZE;

#define dnm(pointer, n, m) (*((pointer)+(n)*dev_MSIZE+(m)))
//#define dev_phi_y(m) (dPhi*((m)-M-1))
#define dev_phi_y(m) (dev_PhiYmin+dPhi*((m)-1))

dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
dim3 dimGrid;

// load data into symbol table
extern "C"
void load_data(void) {
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(E_dc),        &host_E_dc,     sizeof(ffloat)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(E_omega),     &host_E_omega,  sizeof(ffloat)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(omega),       &host_omega,    sizeof(ffloat)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(B),           &host_B,        sizeof(ffloat)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dt),          &host_dt,       sizeof(ffloat)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(M),           &host_M,        sizeof(int)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(N),           &host_N,        sizeof(int)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dPhi),        &host_dPhi,     sizeof(ffloat)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(mu),          &host_mu,       sizeof(ffloat)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(alpha),       &host_alpha,    sizeof(ffloat)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_MSIZE),   &MSIZE,         sizeof(int)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_NSIZE),   &NSIZE,         sizeof(int)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(TMSIZE),      &host_TMSIZE,   sizeof(int)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(bdt),         &host_bdt,      sizeof(ffloat)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nu_tilde),    &host_nu_tilde, sizeof(ffloat)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nu2),         &host_nu2,      sizeof(ffloat)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(nu),          &host_nu,       sizeof(ffloat)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_PhiYmin), &PhiYmin,       sizeof(ffloat)));

  dimGrid.x = (NSIZE+BLOCK_SIZE)/BLOCK_SIZE;
  dimGrid.y = (MP1+BLOCK_SIZE)/BLOCK_SIZE;
} // end of load_data()

__global__ void _step_on_grid_nr(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                                 ffloat *a_next,       ffloat *b_next,
                                 ffloat *a_current_hs, ffloat *b_current_hs,
                                 ffloat t, ffloat t_hs,
                                 ffloat cos_omega_t,   ffloat cos_omega_t_plus_dt)
{
  const int m = threadIdx.x+blockDim.x*blockIdx.x;
  if( m==0 || m > TMSIZE ) { return; }

  // step from (t,t+1/2) to (t+1)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;

  for( int n = 0; n < N; n++ ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    ffloat g = dnm(a_current,n,m)-dnm(b_current,n,m)*mu_t +
      bdt*( dnm(b_current_hs,n+1,m+1) - dnm(b_current_hs,n+1,m-1) - (n < 2 ? 0 : (dnm(b_current_hs,n-1,m+1) - dnm(b_current_hs,n-1,m-1))) );
    ffloat h = dnm(b_current,n,m)+dnm(a_current,n,m)*mu_t +
      bdt*( (n==1?2:1)*(n==0?0:(dnm(a_current_hs,n-1,m+1)-dnm(a_current_hs,n-1,m-1))) - dnm(a_current_hs,n+1,m+1) + dnm(a_current_hs,n+1,m-1) );

    ffloat xi = 1 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next,n,m) = (g - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
      dnm(b_next,n,m) = (g*mu_t_plus_1 + h)/xi;
    }
  }
} // end of _step_on_grid_nr(...)

__global__ void _step_on_half_grid_nr(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                                      ffloat *a_next,       ffloat *b_next,
                                      ffloat *a_current_hs, ffloat *b_current_hs,
                                      ffloat *a_next_hs,    ffloat *b_next_hs,
                                      ffloat t, ffloat t_hs,
                                      ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  const int m = threadIdx.x+blockDim.x*blockIdx.x;
  if( m==0 || m > TMSIZE ) { return; }

  // step from (t+1/2,t+1) to (t+3/2)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;
  for( int n = 0; n < N; n++ ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    ffloat g = dnm(a_current_hs,n,m)-dnm(b_current_hs,n,m)*mu_t +
      bdt*( dnm(b_next,n+1,m+1) - dnm(b_next,n+1,m-1) - (n < 2 ? 0 : (dnm(b_next,n-1,m+1) - dnm(b_next,n-1,m-1))) );
    ffloat h = dnm(b_current_hs,n,m)+dnm(a_current_hs,n,m)*mu_t +
      bdt*( (n==1?2:1)*(n==0?0:(dnm(a_next,n-1,m+1)-dnm(a_next,n-1,m-1))) - dnm(a_next,n+1,m+1) + dnm(a_next,n+1,m-1) );
    ffloat xi = 1 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next_hs,n,m) = (g - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
      dnm(b_next_hs,n,m) = (g*mu_t_plus_1 + h)/xi;
    }
  }
} // end of _step_on_half_grid_nr(...)

__global__ void _step_on_grid_k4(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                              ffloat *a_next,       ffloat *b_next,
                              ffloat *a_current_hs, ffloat *b_current_hs,
                              ffloat t, ffloat t_hs,
                              ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  __shared__ ffloat a_c_forward[TH_PER_BLOCK];
  __shared__ ffloat b_c_forward[TH_PER_BLOCK];

  const int m = threadIdx.x+blockDim.x*blockIdx.x;
  if( m==0 || m > TMSIZE ) { return; }

  // step from (t,t+1/2) to (t+1)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;
  ffloat b_current_hs_n_minus_1_m_plus_1  = 0;
  ffloat b_current_hs_n_minus_1_m_minus_1 = 0;
  ffloat a_current_hs_n_minus_1_m_plus_1  = 0;
  ffloat a_current_hs_n_minus_1_m_minus_1 = 0;
  ffloat b_current_hs_n_plus_1_m_plus_1;
  ffloat b_current_hs_n_plus_1_m_minus_1;
  ffloat a_current_hs_n_plus_1_m_plus_1;
  ffloat a_current_hs_n_plus_1_m_minus_1;

  for( int n = 0; n < N; n += 2 ) {
    ffloat a_center = dnm(a_current,n,m);
    ffloat b_center = dnm(b_current,n,m);
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;

    a_c_forward[threadIdx.x] = dnm(a_current,n+1,m);
    b_c_forward[threadIdx.x] = dnm(b_current,n+1,m);
    __syncthreads();

    b_current_hs_n_plus_1_m_plus_1  = (threadIdx.x==TH_PER_BLOCK_MINUS_ONE)?dnm(b_current_hs,n+1,m+1):b_c_forward[threadIdx.x+1]; // dnm(b_current_hs,n+1,m+1);
    b_current_hs_n_plus_1_m_minus_1 = (threadIdx.x==0)?dnm(b_current_hs,n+1,m-1):b_c_forward[threadIdx.x-1]; // dnm(b_current_hs,n+1,m-1);
    a_current_hs_n_plus_1_m_plus_1  = (threadIdx.x==TH_PER_BLOCK_MINUS_ONE)?dnm(a_current_hs,n+1,m+1):a_c_forward[threadIdx.x+1]; // dnm(a_current_hs,n+1,m+1);
    a_current_hs_n_plus_1_m_minus_1 = (threadIdx.x==0)?dnm(a_current_hs,n+1,m-1):a_c_forward[threadIdx.x-1]; // dnm(a_current_hs,n+1,m-1);
    ffloat g = dt*dnm(a0,n,m)+a_center*nu_tilde-b_center*mu_t +
      bdt*( b_current_hs_n_plus_1_m_plus_1 - b_current_hs_n_plus_1_m_minus_1 - b_current_hs_n_minus_1_m_plus_1 + b_current_hs_n_minus_1_m_minus_1 );
    ffloat h = b_center*nu_tilde+a_center*mu_t +
      bdt*( a_current_hs_n_minus_1_m_plus_1 - a_current_hs_n_minus_1_m_minus_1 - a_current_hs_n_plus_1_m_plus_1 + a_current_hs_n_plus_1_m_minus_1 );

    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
       dnm(b_next,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    }
    b_current_hs_n_minus_1_m_plus_1  = b_current_hs_n_plus_1_m_plus_1;
    b_current_hs_n_minus_1_m_minus_1 = b_current_hs_n_plus_1_m_minus_1;
    a_current_hs_n_minus_1_m_plus_1  = a_current_hs_n_plus_1_m_plus_1;
    a_current_hs_n_minus_1_m_minus_1 = a_current_hs_n_plus_1_m_minus_1;
  }

  b_current_hs_n_minus_1_m_plus_1  = 0;
  b_current_hs_n_minus_1_m_minus_1 = 0;
  a_current_hs_n_minus_1_m_plus_1  = 2*dnm(a_current_hs,0,m+1);
  a_current_hs_n_minus_1_m_minus_1 = 2*dnm(a_current_hs,0,m-1);
  for( int n = 1; n < N; n += 2 ) {
    ffloat a_center = dnm(a_current,n,m);
    ffloat b_center = dnm(b_current,n,m);
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;

    a_c_forward[threadIdx.x] = dnm(a_current_hs,n+1,m);
    b_c_forward[threadIdx.x] = dnm(b_current_hs,n+1,m);
    __syncthreads();

    b_current_hs_n_plus_1_m_plus_1  = (threadIdx.x==TH_PER_BLOCK_MINUS_ONE)?dnm(b_current_hs,n+1,m+1):b_c_forward[threadIdx.x+1]; // dnm(b_current_hs,n+1,m+1);
    b_current_hs_n_plus_1_m_minus_1 = (threadIdx.x==0)?dnm(b_current_hs,n+1,m-1):b_c_forward[threadIdx.x-1]; // dnm(b_current_hs,n+1,m-1);
    a_current_hs_n_plus_1_m_plus_1  = (threadIdx.x==TH_PER_BLOCK_MINUS_ONE)?dnm(a_current_hs,n+1,m+1):a_c_forward[threadIdx.x+1]; // dnm(a_current_hs,n+1,m+1);
    a_current_hs_n_plus_1_m_minus_1 = (threadIdx.x==0)?dnm(a_current_hs,n+1,m-1):a_c_forward[threadIdx.x-1]; // dnm(a_current_hs,n+1,m-1);
    ffloat g = dt*dnm(a0,n,m)+a_center*nu_tilde-b_center*mu_t +
      bdt*( b_current_hs_n_plus_1_m_plus_1 - b_current_hs_n_plus_1_m_minus_1 - b_current_hs_n_minus_1_m_plus_1 + b_current_hs_n_minus_1_m_minus_1);
    ffloat h = b_center*nu_tilde+a_center*mu_t +
      bdt*( a_current_hs_n_minus_1_m_plus_1 - a_current_hs_n_minus_1_m_minus_1 - a_current_hs_n_plus_1_m_plus_1 + a_current_hs_n_plus_1_m_minus_1 );

    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    dnm(b_next,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    b_current_hs_n_minus_1_m_plus_1  = b_current_hs_n_plus_1_m_plus_1;
    b_current_hs_n_minus_1_m_minus_1 = b_current_hs_n_plus_1_m_minus_1;
    a_current_hs_n_minus_1_m_plus_1  = a_current_hs_n_plus_1_m_plus_1;
    a_current_hs_n_minus_1_m_minus_1 = a_current_hs_n_plus_1_m_minus_1;
  }
} // end of _step_on_grid(...)

__global__ void _step_on_half_grid_k4(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                                  ffloat *a_next,       ffloat *b_next,
                                  ffloat *a_current_hs, ffloat *b_current_hs,
                                  ffloat *a_next_hs,    ffloat *b_next_hs,
                                  ffloat t, ffloat t_hs,
                                  ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  __shared__ ffloat a_c_forward[TH_PER_BLOCK];
  __shared__ ffloat b_c_forward[TH_PER_BLOCK];

  const int m = threadIdx.x+blockDim.x*blockIdx.x;
  if( m==0 || m > TMSIZE ) { return; }

  // step from (t+1/2,t+1) to (t+3/2)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;
  ffloat b_next_n_minus_1_m_plus_1  = 0;
  ffloat b_next_n_minus_1_m_minus_1 = 0;
  ffloat a_next_n_minus_1_m_plus_1  = 0;
  ffloat a_next_n_minus_1_m_minus_1 = 0;
  ffloat b_next_n_plus_1_m_plus_1;
  ffloat b_next_n_plus_1_m_minus_1;
  ffloat a_next_n_plus_1_m_plus_1;
  ffloat a_next_n_plus_1_m_minus_1;

  for( int n = 0; n < N; n += 2 ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    ffloat a_center = dnm(a_current_hs,n,m);
    ffloat b_center = dnm(b_current_hs,n,m);

    a_c_forward[threadIdx.x] = dnm(a_next,n+1,m);
    b_c_forward[threadIdx.x] = dnm(b_next,n+1,m);
    __syncthreads();

    b_next_n_plus_1_m_plus_1  = (threadIdx.x==TH_PER_BLOCK_MINUS_ONE)?dnm(b_next,n+1,m+1):b_c_forward[threadIdx.x+1]; // dnm(b_next,n+1,m+1);
    b_next_n_plus_1_m_minus_1 = (threadIdx.x==0)?dnm(b_next,n+1,m-1):b_c_forward[threadIdx.x-1]; // dnm(b_next,n+1,m-1);
    a_next_n_plus_1_m_plus_1  = (threadIdx.x==TH_PER_BLOCK_MINUS_ONE)?dnm(a_next,n+1,m+1):a_c_forward[threadIdx.x+1]; // dnm(a_next,n+1,m+1);
    a_next_n_plus_1_m_minus_1 = (threadIdx.x==0)?dnm(a_next,n+1,m-1):a_c_forward[threadIdx.x-1]; // dnm(a_next,n+1,m-1);
    ffloat g = dt*dnm(a0,n,m)+a_center*nu_tilde-b_center*mu_t +
      bdt*( b_next_n_plus_1_m_plus_1 - b_next_n_plus_1_m_minus_1 - b_next_n_minus_1_m_plus_1 + b_next_n_minus_1_m_minus_1 );
    ffloat h = b_center*nu_tilde+a_center*mu_t +
      bdt*( a_next_n_minus_1_m_plus_1-a_next_n_minus_1_m_minus_1 - a_next_n_plus_1_m_plus_1 + a_next_n_plus_1_m_minus_1 );
    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next_hs,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
      dnm(b_next_hs,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    }
    b_next_n_minus_1_m_plus_1  = b_next_n_plus_1_m_plus_1;
    b_next_n_minus_1_m_minus_1 = b_next_n_plus_1_m_minus_1;
    a_next_n_minus_1_m_plus_1  = a_next_n_plus_1_m_plus_1;
    a_next_n_minus_1_m_minus_1 = a_next_n_plus_1_m_minus_1;
  }

  b_next_n_minus_1_m_plus_1  = 0;
  b_next_n_minus_1_m_minus_1 = 0;
  a_next_n_minus_1_m_plus_1  = 2*dnm(a_next,0,m+1);
  a_next_n_minus_1_m_minus_1 = 2*dnm(a_next,0,m-1);
  for( int n = 1; n < N; n += 2 ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    ffloat a_center = dnm(a_current_hs,n,m);
    ffloat b_center = dnm(b_current_hs,n,m);

    b_next_n_plus_1_m_plus_1  = (threadIdx.x==TH_PER_BLOCK_MINUS_ONE)?dnm(b_next,n+1,m+1):b_c_forward[threadIdx.x+1]; // dnm(b_next,n+1,m+1);
    b_next_n_plus_1_m_minus_1 = (threadIdx.x==0)?dnm(b_next,n+1,m-1):b_c_forward[threadIdx.x-1]; // dnm(b_next,n+1,m-1);
    a_next_n_plus_1_m_plus_1  = (threadIdx.x==TH_PER_BLOCK_MINUS_ONE)?dnm(a_next,n+1,m+1):a_c_forward[threadIdx.x+1]; // dnm(a_next,n+1,m+1);
    a_next_n_plus_1_m_minus_1 = (threadIdx.x==0)?dnm(a_next,n+1,m-1):a_c_forward[threadIdx.x-1]; // dnm(a_next,n+1,m-1);

    ffloat g = dt*dnm(a0,n,m)+a_center*nu_tilde-b_center*mu_t +
      bdt*( b_next_n_plus_1_m_plus_1 - b_next_n_plus_1_m_minus_1 - b_next_n_minus_1_m_plus_1 + b_next_n_minus_1_m_minus_1 );
    ffloat h = b_center*nu_tilde+a_center*mu_t +
      bdt*( a_next_n_minus_1_m_plus_1-a_next_n_minus_1_m_minus_1 - a_next_n_plus_1_m_plus_1 + a_next_n_plus_1_m_minus_1 );
    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next_hs,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    dnm(b_next_hs,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    b_next_n_minus_1_m_plus_1  = b_next_n_plus_1_m_plus_1;
    b_next_n_minus_1_m_minus_1 = b_next_n_plus_1_m_minus_1;
    a_next_n_minus_1_m_plus_1  = a_next_n_plus_1_m_plus_1;
    a_next_n_minus_1_m_minus_1 = a_next_n_plus_1_m_minus_1;
  }
} // end of _step_on_half_grid(...)

__global__ void _step_on_grid_k3(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                              ffloat *a_next,       ffloat *b_next,
                              ffloat *a_current_hs, ffloat *b_current_hs,
                              ffloat t, ffloat t_hs,
                              ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  const int m = threadIdx.x+blockDim.x*blockIdx.x;
  if( m==0 || m > TMSIZE ) { return; }

  // step from (t,t+1/2) to (t+1)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;
  ffloat b_current_hs_n_minus_1_m_plus_1  = 0;
  ffloat b_current_hs_n_minus_1_m_minus_1 = 0;
  ffloat a_current_hs_n_minus_1_m_plus_1  = 0;
  ffloat a_current_hs_n_minus_1_m_minus_1 = 0;
  ffloat b_current_hs_n_plus_1_m_plus_1;
  ffloat b_current_hs_n_plus_1_m_minus_1;
  ffloat a_current_hs_n_plus_1_m_plus_1;
  ffloat a_current_hs_n_plus_1_m_minus_1;
  for( int n = 0; n < N; n += 2 ) {
    ffloat a_center = dnm(a_current,n,m);
    ffloat b_center = dnm(b_current,n,m);
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    b_current_hs_n_plus_1_m_plus_1  = dnm(b_current_hs,n+1,m+1);
    b_current_hs_n_plus_1_m_minus_1 = dnm(b_current_hs,n+1,m-1);
    a_current_hs_n_plus_1_m_plus_1  = dnm(a_current_hs,n+1,m+1);
    a_current_hs_n_plus_1_m_minus_1 = dnm(a_current_hs,n+1,m-1);
    ffloat g = dt*dnm(a0,n,m)+a_center*nu_tilde-b_center*mu_t +
      bdt*( b_current_hs_n_plus_1_m_plus_1 - b_current_hs_n_plus_1_m_minus_1 - b_current_hs_n_minus_1_m_plus_1 + b_current_hs_n_minus_1_m_minus_1 );
    ffloat h = b_center*nu_tilde+a_center*mu_t +
      bdt*( a_current_hs_n_minus_1_m_plus_1 - a_current_hs_n_minus_1_m_minus_1 - a_current_hs_n_plus_1_m_plus_1 + a_current_hs_n_plus_1_m_minus_1 );

    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
       dnm(b_next,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    }
    b_current_hs_n_minus_1_m_plus_1  = b_current_hs_n_plus_1_m_plus_1;
    b_current_hs_n_minus_1_m_minus_1 = b_current_hs_n_plus_1_m_minus_1;
    a_current_hs_n_minus_1_m_plus_1  = a_current_hs_n_plus_1_m_plus_1;
    a_current_hs_n_minus_1_m_minus_1 = a_current_hs_n_plus_1_m_minus_1;
  }
  b_current_hs_n_minus_1_m_plus_1  = 0;
  b_current_hs_n_minus_1_m_minus_1 = 0;
  a_current_hs_n_minus_1_m_plus_1  = 2*dnm(a_current_hs,0,m+1);
  a_current_hs_n_minus_1_m_minus_1 = 2*dnm(a_current_hs,0,m-1);
  for( int n = 1; n < N; n += 2 ) {
    ffloat a_center = dnm(a_current,n,m);
    ffloat b_center = dnm(b_current,n,m);
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    b_current_hs_n_plus_1_m_plus_1  = dnm(b_current_hs,n+1,m+1);
    b_current_hs_n_plus_1_m_minus_1 = dnm(b_current_hs,n+1,m-1);
    a_current_hs_n_plus_1_m_plus_1  = dnm(a_current_hs,n+1,m+1);
    a_current_hs_n_plus_1_m_minus_1 = dnm(a_current_hs,n+1,m-1);
    ffloat g = dt*dnm(a0,n,m)+a_center*nu_tilde-b_center*mu_t +
      bdt*( b_current_hs_n_plus_1_m_plus_1 - b_current_hs_n_plus_1_m_minus_1 - b_current_hs_n_minus_1_m_plus_1 + b_current_hs_n_minus_1_m_minus_1);
    ffloat h = b_center*nu_tilde+a_center*mu_t +
      bdt*( a_current_hs_n_minus_1_m_plus_1 - a_current_hs_n_minus_1_m_minus_1 - a_current_hs_n_plus_1_m_plus_1 + a_current_hs_n_plus_1_m_minus_1 );

    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    dnm(b_next,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    b_current_hs_n_minus_1_m_plus_1  = b_current_hs_n_plus_1_m_plus_1;
    b_current_hs_n_minus_1_m_minus_1 = b_current_hs_n_plus_1_m_minus_1;
    a_current_hs_n_minus_1_m_plus_1  = a_current_hs_n_plus_1_m_plus_1;
    a_current_hs_n_minus_1_m_minus_1 = a_current_hs_n_plus_1_m_minus_1;
  }
} // end of _step_on_grid(...)

__global__ void _step_on_half_grid_k3(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                                  ffloat *a_next,       ffloat *b_next,
                                  ffloat *a_current_hs, ffloat *b_current_hs,
                                  ffloat *a_next_hs,    ffloat *b_next_hs,
                                  ffloat t, ffloat t_hs,
                                  ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  const int m = threadIdx.x+blockDim.x*blockIdx.x;
  if( m==0 || m > TMSIZE ) { return; }

  // step from (t+1/2,t+1) to (t+3/2)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;
  ffloat b_next_n_minus_1_m_plus_1  = 0;
  ffloat b_next_n_minus_1_m_minus_1 = 0;
  ffloat a_next_n_minus_1_m_plus_1  = 0;
  ffloat a_next_n_minus_1_m_minus_1 = 0;
  ffloat b_next_n_plus_1_m_plus_1;
  ffloat b_next_n_plus_1_m_minus_1;
  ffloat a_next_n_plus_1_m_plus_1;
  ffloat a_next_n_plus_1_m_minus_1;

  for( int n = 0; n < N; n += 2 ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    ffloat a_center = dnm(a_current_hs,n,m);
    ffloat b_center = dnm(b_current_hs,n,m);
    b_next_n_plus_1_m_plus_1  = dnm(b_next,n+1,m+1);
    b_next_n_plus_1_m_minus_1 = dnm(b_next,n+1,m-1);
    a_next_n_plus_1_m_plus_1  = dnm(a_next,n+1,m+1);
    a_next_n_plus_1_m_minus_1 = dnm(a_next,n+1,m-1);
    ffloat g = dt*dnm(a0,n,m)+a_center*nu_tilde-b_center*mu_t +
      bdt*( b_next_n_plus_1_m_plus_1 - b_next_n_plus_1_m_minus_1 - b_next_n_minus_1_m_plus_1 + b_next_n_minus_1_m_minus_1 );
    ffloat h = b_center*nu_tilde+a_center*mu_t +
      bdt*( a_next_n_minus_1_m_plus_1-a_next_n_minus_1_m_minus_1 - a_next_n_plus_1_m_plus_1 + a_next_n_plus_1_m_minus_1 );
    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next_hs,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
      dnm(b_next_hs,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    }
    b_next_n_minus_1_m_plus_1  = b_next_n_plus_1_m_plus_1;
    b_next_n_minus_1_m_minus_1 = b_next_n_plus_1_m_minus_1;
    a_next_n_minus_1_m_plus_1  = a_next_n_plus_1_m_plus_1;
    a_next_n_minus_1_m_minus_1 = a_next_n_plus_1_m_minus_1;
  }

  b_next_n_minus_1_m_plus_1  = 0;
  b_next_n_minus_1_m_minus_1 = 0;
  a_next_n_minus_1_m_plus_1  = 2*dnm(a_next,0,m+1);
  a_next_n_minus_1_m_minus_1 = 2*dnm(a_next,0,m-1);
  for( int n = 1; n < N; n += 2 ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    ffloat a_center = dnm(a_current_hs,n,m);
    ffloat b_center = dnm(b_current_hs,n,m);
    b_next_n_plus_1_m_plus_1  = dnm(b_next,n+1,m+1);
    b_next_n_plus_1_m_minus_1 = dnm(b_next,n+1,m-1);
    a_next_n_plus_1_m_plus_1  = dnm(a_next,n+1,m+1);
    a_next_n_plus_1_m_minus_1 = dnm(a_next,n+1,m-1);
    ffloat g = dt*dnm(a0,n,m)+a_center*nu_tilde-b_center*mu_t +
      bdt*( b_next_n_plus_1_m_plus_1 - b_next_n_plus_1_m_minus_1 - b_next_n_minus_1_m_plus_1 + b_next_n_minus_1_m_minus_1 );
    ffloat h = b_center*nu_tilde+a_center*mu_t +
      bdt*( a_next_n_minus_1_m_plus_1-a_next_n_minus_1_m_minus_1 - a_next_n_plus_1_m_plus_1 + a_next_n_plus_1_m_minus_1 );
    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next_hs,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    dnm(b_next_hs,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    b_next_n_minus_1_m_plus_1  = b_next_n_plus_1_m_plus_1;
    b_next_n_minus_1_m_minus_1 = b_next_n_plus_1_m_minus_1;
    a_next_n_minus_1_m_plus_1  = a_next_n_plus_1_m_plus_1;
    a_next_n_minus_1_m_minus_1 = a_next_n_plus_1_m_minus_1;
  }
} // end of _step_on_half_grid(...)

__global__ void _step_on_grid_k6(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                              ffloat *a_next,       ffloat *b_next,
                              ffloat *a_current_hs, ffloat *b_current_hs,
                              ffloat t, ffloat t_hs,
                              ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  const int m = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if( m > TMSIZE || n >= N ) { return; }

  __shared__ ffloat a_c[(BLOCK_SIZE+2)*(BLOCK_SIZE+2)];
  __shared__ ffloat b_c[(BLOCK_SIZE+2)*(BLOCK_SIZE+2)];
  a_c[BLOCK_SIZE*threadIdx.x+threadIdx.y] = dnm(a_current_hs,n,m);
  b_c[BLOCK_SIZE*threadIdx.x+threadIdx.y] = dnm(b_current_hs,n,m);
  __syncthreads();

  // step from (t,t+1/2) to (t+1)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;

  //for( int n = 0; n < N; n++ ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;

    //ffloat g = dt*dnm(a0,n,m)+dnm(a_current,n,m)*nu_tilde-dnm(b_current,n,m)*mu_t +
    //  bdt*( dnm(b_current_hs,n+1,m+1) - dnm(b_current_hs,n+1,m-1) - (n < 2 ? 0 : (dnm(b_current_hs,n-1,m+1) - dnm(b_current_hs,n-1,m-1))) );
    ffloat g = dt*dnm(a0,n,m)+dnm(a_current,n,m)*nu_tilde-dnm(b_current,n,m)*mu_t +
      bdt*( ((threadIdx.x<BLOCK_SIZE_M1 && threadIdx.y<BLOCK_SIZE_M1)?b_c[BLOCK_SIZE*(threadIdx.x+1)+threadIdx.y+1]:dnm(b_current_hs,n+1,m+1))- 
	    ((threadIdx.x<BLOCK_SIZE_M1 && threadIdx.y!=0)?b_c[BLOCK_SIZE*(threadIdx.x+1)+threadIdx.y-1]:dnm(b_current_hs,n+1,m-1)) - 
	    (n < 2 ? 0 : (
			  ((threadIdx.x!=0 && threadIdx.y!=BLOCK_SIZE_M1)?b_c[BLOCK_SIZE*(threadIdx.x-1)+threadIdx.y+1]:dnm(b_current_hs,n-1,m+1)) - 
			  ((threadIdx.x!=0 && threadIdx.y!=0)?b_c[BLOCK_SIZE*(threadIdx.x-1)+threadIdx.y-1]:dnm(b_current_hs,n-1,m-1))
			  )) );

    //ffloat h = dnm(b_current,n,m)*nu_tilde+dnm(a_current,n,m)*mu_t +
    //  bdt*( (n==1?2:1)*(n==0?0:(dnm(a_current_hs,n-1,m+1)-dnm(a_current_hs,n-1,m-1))) - dnm(a_current_hs,n+1,m+1) + dnm(a_current_hs,n+1,m-1) );

    ffloat h = dnm(b_current,n,m)*nu_tilde+dnm(a_current,n,m)*mu_t +
      bdt*( (n==1?2:1)*(n==0?0:(
				((threadIdx.x!=0 && threadIdx.y!=BLOCK_SIZE_M1)?a_c[BLOCK_SIZE*(threadIdx.x-1)+threadIdx.y+1]:dnm(a_current_hs,n-1,m+1)) -
				((threadIdx.x!=0 && threadIdx.y!=0)?a_c[BLOCK_SIZE*(threadIdx.x-1)+threadIdx.y-1]:dnm(a_current_hs,n-1,m-1))
				)) - 
	    ((threadIdx.x<BLOCK_SIZE_M1 && threadIdx.y<BLOCK_SIZE_M1)?a_c[BLOCK_SIZE*(threadIdx.x+1)+threadIdx.y+1]:dnm(a_current_hs,n+1,m+1)) + 
	    ((threadIdx.x<BLOCK_SIZE_M1 && threadIdx.y!=0)?a_c[BLOCK_SIZE*(threadIdx.x+1)+threadIdx.y-1]:dnm(a_current_hs,n+1,m-1)));

    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
      dnm(b_next,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    }
  //}
} // end of _step_on_grid_k6(...)

__global__ void _step_on_half_grid_k6(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                                  ffloat *a_next,       ffloat *b_next,
                                  ffloat *a_current_hs, ffloat *b_current_hs,
                                  ffloat *a_next_hs,    ffloat *b_next_hs,
                                  ffloat t, ffloat t_hs,
                                  ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  const int m = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if( m > TMSIZE || n >= N ) { return; }

  __shared__ ffloat a_c[(BLOCK_SIZE+2)*(BLOCK_SIZE+2)];
  __shared__ ffloat b_c[(BLOCK_SIZE+2)*(BLOCK_SIZE+2)];
  a_c[BLOCK_SIZE*threadIdx.x+threadIdx.y] = dnm(a_next,n,m);
  b_c[BLOCK_SIZE*threadIdx.x+threadIdx.y] = dnm(b_next,n,m);
  __syncthreads();

  // step from (t+1/2,t+1) to (t+3/2)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;
  //for( int n = 0; n < N; n++ ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    ffloat g = dt*dnm(a0,n,m)+dnm(a_current_hs,n,m)*nu_tilde-dnm(b_current_hs,n,m)*mu_t +

      bdt*( ((threadIdx.x<BLOCK_SIZE_M1 && threadIdx.y<BLOCK_SIZE_M1)?b_c[BLOCK_SIZE*(threadIdx.x+1)+threadIdx.y+1]:dnm(b_next,n+1,m+1))- 
	    ((threadIdx.x<BLOCK_SIZE_M1 && threadIdx.y!=0)?b_c[BLOCK_SIZE*(threadIdx.x+1)+threadIdx.y-1]:dnm(b_next,n+1,m-1)) - 
	    (n < 2 ? 0 : (
			  ((threadIdx.x!=0 && threadIdx.y!=BLOCK_SIZE_M1)?b_c[BLOCK_SIZE*(threadIdx.x-1)+threadIdx.y+1]:dnm(b_next,n-1,m+1)) - 
			  ((threadIdx.x!=0 && threadIdx.y!=0)?b_c[BLOCK_SIZE*(threadIdx.x-1)+threadIdx.y-1]:dnm(b_next,n-1,m-1))
			  )) );


    ffloat h = dnm(b_current_hs,n,m)*nu_tilde+dnm(a_current_hs,n,m)*mu_t +
      bdt*( (n==1?2:1)*(n==0?0:(
				((threadIdx.x!=0 && threadIdx.y!=BLOCK_SIZE_M1)?a_c[BLOCK_SIZE*(threadIdx.x-1)+threadIdx.y+1]:dnm(a_next,n-1,m+1)) -
				((threadIdx.x!=0 && threadIdx.y!=0)?a_c[BLOCK_SIZE*(threadIdx.x-1)+threadIdx.y-1]:dnm(a_next,n-1,m-1))
				)) - 
	    ((threadIdx.x<BLOCK_SIZE_M1 && threadIdx.y<BLOCK_SIZE_M1)?a_c[BLOCK_SIZE*(threadIdx.x+1)+threadIdx.y+1]:dnm(a_next,n+1,m+1)) + 
	    ((threadIdx.x<BLOCK_SIZE_M1 && threadIdx.y!=0)?a_c[BLOCK_SIZE*(threadIdx.x+1)+threadIdx.y-1]:dnm(a_next,n+1,m-1)));

    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next_hs,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
      dnm(b_next_hs,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    }
  //}
} // end of _step_on_half_grid_k6(...)

__global__ void _step_on_grid_k5(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                              ffloat *a_next,       ffloat *b_next,
                              ffloat *a_current_hs, ffloat *b_current_hs,
                              ffloat t, ffloat t_hs,
                              ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  const int m = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if( m > TMSIZE || n >= N ) { return; }

  // step from (t,t+1/2) to (t+1)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;

  //for( int n = 0; n < N; n++ ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    ffloat g = dt*dnm(a0,n,m)+dnm(a_current,n,m)*nu_tilde-dnm(b_current,n,m)*mu_t +
      bdt*( dnm(b_current_hs,n+1,m+1) - dnm(b_current_hs,n+1,m-1) - (n < 2 ? 0 : (dnm(b_current_hs,n-1,m+1) - dnm(b_current_hs,n-1,m-1))) );
    ffloat h = dnm(b_current,n,m)*nu_tilde+dnm(a_current,n,m)*mu_t +
      bdt*( (n==1?2:1)*(n==0?0:(dnm(a_current_hs,n-1,m+1)-dnm(a_current_hs,n-1,m-1))) - dnm(a_current_hs,n+1,m+1) + dnm(a_current_hs,n+1,m-1) );

    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
      dnm(b_next,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    }
  //}
} // end of _step_on_grid_k1(...)

__global__ void _step_on_half_grid_k5(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                                  ffloat *a_next,       ffloat *b_next,
                                  ffloat *a_current_hs, ffloat *b_current_hs,
                                  ffloat *a_next_hs,    ffloat *b_next_hs,
                                  ffloat t, ffloat t_hs,
                                  ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  const int m = blockIdx.y * blockDim.y + threadIdx.y + 1;
  const int n = blockIdx.x * blockDim.x + threadIdx.x;
  if( m > TMSIZE || n >= N ) { return; }

  // step from (t+1/2,t+1) to (t+3/2)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;
  //for( int n = 0; n < N; n++ ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    ffloat g = dt*dnm(a0,n,m)+dnm(a_current_hs,n,m)*nu_tilde-dnm(b_current_hs,n,m)*mu_t +
      bdt*( dnm(b_next,n+1,m+1) - dnm(b_next,n+1,m-1) - (n < 2 ? 0 : (dnm(b_next,n-1,m+1) - dnm(b_next,n-1,m-1))) );
    ffloat h = dnm(b_current_hs,n,m)*nu_tilde+dnm(a_current_hs,n,m)*mu_t +
      bdt*( (n==1?2:1)*(n==0?0:(dnm(a_next,n-1,m+1)-dnm(a_next,n-1,m-1))) - dnm(a_next,n+1,m+1) + dnm(a_next,n+1,m-1) );
    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next_hs,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
      dnm(b_next_hs,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    }
  //}
} // end of _step_on_half_grid_k5(...)

__global__ void _step_on_grid_k1(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                              ffloat *a_next,       ffloat *b_next,
                              ffloat *a_current_hs, ffloat *b_current_hs,
                              ffloat t, ffloat t_hs,
                              ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  const int m = threadIdx.x+blockDim.x*blockIdx.x;
  if( m==0 || m > TMSIZE ) { return; }

  // step from (t,t+1/2) to (t+1)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;

  for( int n = 0; n < N; n++ ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    ffloat g = dt*dnm(a0,n,m)+dnm(a_current,n,m)*nu_tilde-dnm(b_current,n,m)*mu_t +
      bdt*( dnm(b_current_hs,n+1,m+1) - dnm(b_current_hs,n+1,m-1) - (n < 2 ? 0 : (dnm(b_current_hs,n-1,m+1) - dnm(b_current_hs,n-1,m-1))) );
    ffloat h = dnm(b_current,n,m)*nu_tilde+dnm(a_current,n,m)*mu_t +
      bdt*( (n==1?2:1)*(n==0?0:(dnm(a_current_hs,n-1,m+1)-dnm(a_current_hs,n-1,m-1))) - dnm(a_current_hs,n+1,m+1) + dnm(a_current_hs,n+1,m-1) );

    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
      dnm(b_next,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    }
  }
} // end of _step_on_grid_k1(...)

__global__ void _step_on_half_grid_k1(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                                  ffloat *a_next,       ffloat *b_next,
                                  ffloat *a_current_hs, ffloat *b_current_hs,
                                  ffloat *a_next_hs,    ffloat *b_next_hs,
                                  ffloat t, ffloat t_hs,
                                  ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  const int m = threadIdx.x+blockDim.x*blockIdx.x;
  if( m==0 || m > TMSIZE ) { return; }

  // step from (t+1/2,t+1) to (t+3/2)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;
  for( int n = 0; n < N; n++ ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    ffloat g = dt*dnm(a0,n,m)+dnm(a_current_hs,n,m)*nu_tilde-dnm(b_current_hs,n,m)*mu_t +
      bdt*( dnm(b_next,n+1,m+1) - dnm(b_next,n+1,m-1) - (n < 2 ? 0 : (dnm(b_next,n-1,m+1) - dnm(b_next,n-1,m-1))) );
    ffloat h = dnm(b_current_hs,n,m)*nu_tilde+dnm(a_current_hs,n,m)*mu_t +
      bdt*( (n==1?2:1)*(n==0?0:(dnm(a_next,n-1,m+1)-dnm(a_next,n-1,m-1))) - dnm(a_next,n+1,m+1) + dnm(a_next,n+1,m-1) );
    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next_hs,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
      dnm(b_next_hs,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    }
  }
} // end of _step_on_half_grid_k1(...)

__global__ void _step_on_grid_k2(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                              ffloat *a_next,       ffloat *b_next,
                              ffloat *a_current_hs, ffloat *b_current_hs,
                              ffloat t, ffloat t_hs,
                              ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  const int m = threadIdx.x+blockDim.x*blockIdx.x;
  if( m==0 || m > TMSIZE ) { return; }

  // step from (t,t+1/2) to (t+1)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;

  #pragma unroll 1
  for( int n = 0; n < N; n++ ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    ffloat g = dt*dnm(a0,n,m)+dnm(a_current,n,m)*nu_tilde-dnm(b_current,n,m)*mu_t +
      bdt*( dnm(b_current_hs,n+1,m+1) - dnm(b_current_hs,n+1,m-1) - (n < 2 ? 0 : (dnm(b_current_hs,n-1,m+1) - dnm(b_current_hs,n-1,m-1))) );
    ffloat h = dnm(b_current,n,m)*nu_tilde+dnm(a_current,n,m)*mu_t +
      bdt*( (n==1?2:1)*(n==0?0:(dnm(a_current_hs,n-1,m+1)-dnm(a_current_hs,n-1,m-1))) - dnm(a_current_hs,n+1,m+1) + dnm(a_current_hs,n+1,m-1) );

    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
      dnm(b_next,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    }
  }
} // end of _step_on_grid_k2(...)

__global__ void _step_on_half_grid_k2(ffloat *a0, ffloat *a_current,    ffloat *b_current,
                                  ffloat *a_next,       ffloat *b_next,
                                  ffloat *a_current_hs, ffloat *b_current_hs,
                                  ffloat *a_next_hs,    ffloat *b_next_hs,
                                  ffloat t, ffloat t_hs,
                                  ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  const int m = threadIdx.x+blockDim.x*blockIdx.x;
  if( m==0 || m > TMSIZE ) { return; }

  // step from (t+1/2,t+1) to (t+3/2)
  ffloat mu_t_part = (E_dc + E_omega*cos_omega_t+B*dev_phi_y(m))*dt/2;
  ffloat mu_t_plus_1_part = (E_dc + E_omega*cos_omega_t_plus_dt+B*dev_phi_y(m))*dt/2;
  #pragma unroll 1
  for( int n = 0; n < N; n++ ) {
    ffloat mu_t = n*mu_t_part;
    ffloat mu_t_plus_1 = n*mu_t_plus_1_part;
    ffloat g = dt*dnm(a0,n,m)+dnm(a_current_hs,n,m)*nu_tilde-dnm(b_current_hs,n,m)*mu_t +
      bdt*( dnm(b_next,n+1,m+1) - dnm(b_next,n+1,m-1) - (n < 2 ? 0 : (dnm(b_next,n-1,m+1) - dnm(b_next,n-1,m-1))) );
    ffloat h = dnm(b_current_hs,n,m)*nu_tilde+dnm(a_current_hs,n,m)*mu_t +
      bdt*( (n==1?2:1)*(n==0?0:(dnm(a_next,n-1,m+1)-dnm(a_next,n-1,m-1))) - dnm(a_next,n+1,m+1) + dnm(a_next,n+1,m-1) );
    ffloat xi = nu2 + mu_t_plus_1*mu_t_plus_1;
    dnm(a_next_hs,n,m) = (g*nu - h*mu_t_plus_1)/xi;
    if( n > 0 ) {
      dnm(b_next_hs,n,m) = (g*mu_t_plus_1 + h*nu)/xi;
    }
  }
} // end of _step_on_half_grid_k2(...)

__global__ void av_gpu_parallel(ffloat *a, ffloat *b, ffloat *av_data, ffloat t) {
  //threadIdx.x;
  //blockIdx.x;
  //blockDim.x; // number of threads per block

  __shared__ ffloat v_dr_acc[PPP];
  __shared__ ffloat v_y_acc[PPP];
  __shared__ ffloat m_over_m_x_inst_acc[PPP];

  int thid = threadIdx.x;
  v_dr_acc[thid]            = 0;
  v_y_acc[thid]             = 0;
  m_over_m_x_inst_acc[thid] = 0;
  for( int i = thid+1; i < TMSIZE; i += PPP ) {
    v_dr_acc[thid]            += dnm(b,1,i)*dPhi;
    v_y_acc[thid]             += dnm(a,0,i)*dev_phi_y(i)*dPhi;
    m_over_m_x_inst_acc[thid] += dnm(a,1,i)*dPhi;
  }

  __syncthreads();

  //for(int delta = PPP/2; delta > 0; delta /= 2 ) {
  //int delta = PPP/2;
  //  for( int i = thid; i < delta; i++ ) {
  //    v_dr_acc[i]            += v_dr_acc[i+delta];
  //    v_y_acc[i]             += v_y_acc[i+delta];
  //    m_over_m_x_inst_acc[i] += m_over_m_x_inst_acc[i+delta];
  //  }
  //  __syncthreads();
    //}
    //__syncthreads();

  if( thid == 0 ) {
    int av_count = av_data[0] + 1;
    ffloat v_dr_inst = 0; ffloat v_y_inst = 0; ffloat m_over_m_x_inst = 0;
    for( int m = 0; m < PPP; m++ ) {
      v_dr_inst += v_dr_acc[m];
      v_y_inst  += v_y_acc[m];
      m_over_m_x_inst += m_over_m_x_inst_acc[thid];
    }
    //ffloat v_dr_inst = v_dr_acc[0]; ffloat v_y_inst = v_y_acc[0]; ffloat m_over_m_x_inst = m_over_m_x_inst_acc[0];
    //v_dr_av = v_dr_av+(v_dr_inst-v_dr_av)/av_count;
    av_data[1] += (v_dr_inst-av_data[1])/av_count; // av_data[1] holds v_dr_av

    //v_y_av = v_y_av+(v_y_inst-v_y_av)/av_count;
    av_data[2] += (v_y_inst-av_data[2])/av_count; // av_data[2] holds v_y_av

    //m_over_m_x_av = m_over_m_x_av+(m_over_m_x_inst-m_over_m_x_av)/av_count;
    av_data[3] += (m_over_m_x_inst-av_data[3])/av_count; // av_data[3] holds m_over_m_x_av

    //A += cos(omega*t)*v_dr_inst*dt;
    av_data[4] += cos(omega*t)*v_dr_inst*dt; // av_data[4] holds absorption A
    av_data[5] += sin(omega*t)*v_dr_inst*dt; // av_data[5] holds sin absorption A

    av_data[0] += 1;
  }
} // end of av_gpu_parallel(...)

__global__ void av_gpu(ffloat *a, ffloat *b, ffloat *av_data, ffloat t) {
  int av_count = av_data[0] + 1;

  ffloat v_dr_inst = 0; ffloat v_y_inst = 0; ffloat m_over_m_x_inst = 0;
  for( int m = 1; m < TMSIZE; m++ ) {
    v_dr_inst += dnm(b,1,m)*dPhi;
    v_y_inst  += dnm(a,0,m)*dev_phi_y(m)*dPhi;
    m_over_m_x_inst += dnm(a,1,m)*dPhi;
  }

  //v_dr_av = v_dr_av+(v_dr_inst-v_dr_av)/av_count;
  av_data[1] += (v_dr_inst-av_data[1])/av_count; // av_data[1] holds v_dr_av

  //v_y_av = v_y_av+(v_y_inst-v_y_av)/av_count;
  av_data[2] += (v_y_inst-av_data[2])/av_count; // av_data[2] holds v_y_av

  //m_over_m_x_av = m_over_m_x_av+(m_over_m_x_inst-m_over_m_x_av)/av_count;
  av_data[3] += (m_over_m_x_inst-av_data[3])/av_count; // av_data[3] holds m_over_m_x_av

  //A += cos(omega*t)*v_dr_inst*dt;
  av_data[4] += cos(omega*t)*v_dr_inst*dt; // av_data[4] holds absorption A
  av_data[5] += sin(omega*t)*v_dr_inst*dt; // av_data[4] holds sin absorption A

  av_data[0] += 1;
} // end of av_gpu(...)

extern "C"
void step_on_grid(int blocks, ffloat *a0, ffloat *a_current,    ffloat *b_current,
                  ffloat *a_next,       ffloat *b_next,
                  ffloat *a_current_hs, ffloat *b_current_hs,
                  ffloat t, ffloat t_hs, ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
#if BLTZM_KERNEL == 1
  _step_on_grid_k1<<<blocks,TH_PER_BLOCK>>>(a0, a_current, b_current, a_next, b_next,
                                         a_current_hs, b_current_hs,
                                         t, t_hs, cos_omega_t, cos_omega_t_plus_dt);
#elif BLTZM_KERNEL == 2
  _step_on_grid_k2<<<blocks,TH_PER_BLOCK>>>(a0, a_current, b_current, a_next, b_next,
                                         a_current_hs, b_current_hs,
                                         t, t_hs, cos_omega_t, cos_omega_t_plus_dt);
#elif BLTZM_KERNEL == 3
  _step_on_grid_k3<<<blocks,TH_PER_BLOCK>>>(a0, a_current, b_current, a_next, b_next,
                                         a_current_hs, b_current_hs,
                                         t, t_hs, cos_omega_t, cos_omega_t_plus_dt);
#elif BLTZM_KERNEL == 4
  _step_on_grid_k4<<<blocks,TH_PER_BLOCK>>>(a0, a_current, b_current, a_next, b_next,
                                         a_current_hs, b_current_hs,
                                         t, t_hs, cos_omega_t, cos_omega_t_plus_dt);
#elif BLTZM_KERNEL == 5
  _step_on_grid_k5<<<dimGrid, dimBlock>>>(a0, a_current, b_current, a_next, b_next,
                                         a_current_hs, b_current_hs,
                                         t, t_hs, cos_omega_t, cos_omega_t_plus_dt);

#elif BLTZM_KERNEL == 6
  _step_on_grid_k6<<<dimGrid, dimBlock>>>(a0, a_current, b_current, a_next, b_next,
                                         a_current_hs, b_current_hs,
                                         t, t_hs, cos_omega_t, cos_omega_t_plus_dt);

#else 
  _step_on_grid<<<blocks,TH_PER_BLOCK>>>(a0, a_current, b_current, a_next, b_next,
                                         a_current_hs, b_current_hs,
                                         t, t_hs, cos_omega_t, cos_omega_t_plus_dt);
#endif
}

extern "C"
void step_on_half_grid(int blocks, ffloat *a0, ffloat *a_current,    ffloat *b_current,
                       ffloat *a_next,       ffloat *b_next,
                       ffloat *a_current_hs, ffloat *b_current_hs,
                       ffloat *a_next_hs, ffloat *b_next_hs,
                       ffloat t, ffloat t_hs, ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
#if BLTZM_KERNEL == 1
  _step_on_half_grid_k1<<<blocks,TH_PER_BLOCK>>>(a0, a_current, b_current, a_next, b_next,
                                              a_current_hs, b_current_hs, a_next_hs, b_next_hs,
                                              t, t_hs, cos_omega_t, cos_omega_t_plus_dt);
#elif BLTZM_KERNEL == 2
  _step_on_half_grid_k2<<<blocks,TH_PER_BLOCK>>>(a0, a_current, b_current, a_next, b_next,
                                              a_current_hs, b_current_hs, a_next_hs, b_next_hs,
                                              t, t_hs, cos_omega_t, cos_omega_t_plus_dt);
#elif BLTZM_KERNEL == 3
  _step_on_half_grid_k3<<<blocks,TH_PER_BLOCK>>>(a0, a_current, b_current, a_next, b_next,
                                              a_current_hs, b_current_hs, a_next_hs, b_next_hs,
                                              t, t_hs, cos_omega_t, cos_omega_t_plus_dt);
#elif BLTZM_KERNEL == 4
  _step_on_half_grid_k4<<<blocks,TH_PER_BLOCK>>>(a0, a_current, b_current, a_next, b_next,
                                              a_current_hs, b_current_hs, a_next_hs, b_next_hs,
                                              t, t_hs, cos_omega_t, cos_omega_t_plus_dt);
#elif BLTZM_KERNEL == 5

  //printf("dimBlock(%d,%d)\n", dimBlock.x, dimBlock.y);
  //printf("dimGrid(%d,%d)\n", dimGrid.x, dimGrid.y);
  //printf("%d, %d, %d\n", MP1, BLOCK_SIZE, ((MP1+BLOCK_SIZE)/BLOCK_SIZE));
  _step_on_half_grid_k5<<<dimGrid, dimBlock>>>(a0, a_current, b_current, a_next, b_next,
						 a_current_hs, b_current_hs, a_next_hs, b_next_hs,
						 t, t_hs, cos_omega_t, cos_omega_t_plus_dt);

#elif BLTZM_KERNEL == 6
  _step_on_half_grid_k6<<<dimGrid, dimBlock>>>(a0, a_current, b_current, a_next, b_next,
						 a_current_hs, b_current_hs, a_next_hs, b_next_hs,
						 t, t_hs, cos_omega_t, cos_omega_t_plus_dt);

#else
  _step_on_half_grid<<<blocks,TH_PER_BLOCK>>>(a0, a_current, b_current, a_next, b_next,
                                              a_current_hs, b_current_hs, a_next_hs, b_next_hs,
                                              t, t_hs, cos_omega_t, cos_omega_t_plus_dt);
#endif
}

extern "C"
void step_on_grid_nr(int blocks, ffloat *a0, ffloat *a_current,    ffloat *b_current,
                  ffloat *a_next,       ffloat *b_next,
                  ffloat *a_current_hs, ffloat *b_current_hs,
                  ffloat t, ffloat t_hs, ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  _step_on_grid_nr<<<blocks,TH_PER_BLOCK>>>(a0, a_current, b_current, a_next, b_next,
                                         a_current_hs, b_current_hs,
                                         t, t_hs, cos_omega_t, cos_omega_t_plus_dt);
}

extern "C"
void step_on_half_grid_nr(int blocks, ffloat *a0, ffloat *a_current,    ffloat *b_current,
                       ffloat *a_next,       ffloat *b_next,
                       ffloat *a_current_hs, ffloat *b_current_hs,
                       ffloat *a_next_hs, ffloat *b_next_hs,
                       ffloat t, ffloat t_hs, ffloat cos_omega_t, ffloat cos_omega_t_plus_dt)
{
  _step_on_half_grid_nr<<<blocks,TH_PER_BLOCK>>>(a0, a_current, b_current, a_next, b_next,
                                              a_current_hs, b_current_hs, a_next_hs, b_next_hs,
                                              t, t_hs, cos_omega_t, cos_omega_t_plus_dt);
}

extern "C"
void av(int blocks, ffloat *a, ffloat *b, ffloat *av_data, ffloat t) {
  av_gpu_parallel<<<1,PPP>>>(a, b, av_data, t);
  //av_gpu<<<1,1>>>(a, b, av_data, t);
}
